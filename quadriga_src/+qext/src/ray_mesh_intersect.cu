#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cmath>
#include "qd_mesh_functions.h"
#include "ray_mesh_intersect.h"
using namespace std;

//Define an assert style handler function and wrapper macro to check for errors in runtime API code
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
    if (code != hipSuccess)
        cout << "GPUassert: '" << hipGetErrorString(code) << "' in " << file << ":" << line << endl;
}

// Block dimensions must be declared statically to correcly initiate shared memory
#define BLOCK_SIZE_RayMesh 256 // Block size for "RayMeshIntersect"
#define BLOCK_SIZE_RayRx 512   // Block size for "RayRxIntersect"
#define SERIAL_RAYS_RayRx 32   // Number of rays to be processed in serial in a Kernel

// KERNEL: Calculate "dest" = "dest" - "orig"
__global__ void DestMinusOrig(float *Ox, float *Oy, float *Oz, float *Dx, float *Dy, float *Dz)
{
    unsigned int i_ray = blockIdx.x * blockDim.x + threadIdx.x; // Ray index
    Dx[i_ray] = Dx[i_ray] - Ox[i_ray];
    Dy[i_ray] = Dy[i_ray] - Oy[i_ray];
    Dz[i_ray] = Dz[i_ray] - Oz[i_ray];
}

// KERNEL: Test if a ray hits a mesh element
__global__ void RayMeshIntersect(float *v1x, float *v1y, float *v1z, float *v2x, float *v2y, float *v2z,
                                 float *v3x, float *v3y, float *v3z, float *Ox, float *Oy, float *Oz,
                                 float *Dx, float *Dy, float *Dz, float *Wx, float *Wy, unsigned int *Wz, unsigned int *hit_cnt,
                                 float *Wout, unsigned int no_hit_W)
{
    // Input variables:
    //  v1x - v3z       Mesh coordinates, vectors of lenght [no_mesh]
    //  Ox, Oy, Oz      Ray origin, vectors of length [no_rays]
    //  Dx, Dy, Dz      Ray destinations, vectors of length [no_rays]
    //  no_hit_W        Number of interactions to be retunrned in the output

    // Output variables:
    //  Wx              Minimum normalized distance, values between 0 and 1, vector of lenght [no_ray_chnk * no_mesh_blk]
    //  Wy              Maximum normalized distance, values between 0 and 1, vector of lenght [no_ray_chnk * no_mesh_blk]
    //  Wz              Index of the first mesh element that was hit by the ray, vector of length [no_ray_chnk * no_mesh_blk]
    //  hit_cnt         Number of mesh indersections, vector of length [no_ray]
    //  Wout            List of normalized distances, values between 0 and 1, vector of lenght [no_ray * no_hit_W]

    // Get the current ray and mesh element
    unsigned int i_ray = blockIdx.x;
    unsigned int i_mesh = blockIdx.y * BLOCK_SIZE_RayMesh + threadIdx.x;

    // Shared variable to locally store results and ray data of one block
    __shared__ float Wshared[BLOCK_SIZE_RayMesh];
    __shared__ unsigned int Windex[BLOCK_SIZE_RayMesh];
    __shared__ unsigned int no_hit;
    __shared__ float k_Dx, k_Dy, k_Dz, k_Ox, k_Oy, k_Oz;

    // First thread initializs the local hit counter in shared memory
    if (threadIdx.x == 0)
        no_hit = 0;

    // Read one ray coordinate per warp and store it in shared memory
    if (threadIdx.x == 32)
        k_Dx = Dx[i_ray];
    if (threadIdx.x == 64)
        k_Dy = Dy[i_ray];
    if (threadIdx.x == 96)
        k_Dz = Dz[i_ray];
    if (threadIdx.x == 128)
        k_Ox = Ox[i_ray];
    if (threadIdx.x == 160)
        k_Oy = Oy[i_ray];
    if (threadIdx.x == 192)
        k_Oz = Oz[i_ray];

    // Synchronize so that all threads have access to the shared data
    __syncthreads();

    // Calculate Vector from V1 to O
    float Tx = k_Ox - v1x[i_mesh];
    float Ty = k_Oy - v1y[i_mesh];
    float Tz = k_Oz - v1z[i_mesh];

    // Read edges V1-V2 and and V1-V3 from global memory
    float k_v2x = v2x[i_mesh];
    float k_v2z = v2z[i_mesh];
    float k_v2y = v2y[i_mesh];
    float k_v3x = v3x[i_mesh];
    float k_v3y = v3y[i_mesh];
    float k_v3z = v3z[i_mesh];

    // Calculate 1st barycentric coordinate (gU)
    float PQ = k_v3z * k_Dy - k_v3y * k_Dz;
    float R0 = 3.74e-23;
    float DT = R0 + k_v2x * PQ;
    float U = Tx * PQ;

    PQ = k_v3x * k_Dz - k_v3z * k_Dx;
    DT = DT + k_v2y * PQ;
    U = U + Ty * PQ;
    PQ = k_v3y * k_Dx - k_v3x * k_Dy;
    DT = DT + k_v2z * PQ;
    U = U + Tz * PQ;

    DT = 1 / DT;
    U = U * DT;

    // Calculate and 2nd barycentric coordinate (gV)
    // Calculate and normalized line intersect position (gW)
    PQ = k_v2z * Ty - k_v2y * Tz;
    float V = PQ * k_Dx;
    float W = PQ * k_v3x;

    PQ = k_v2x * Tz - k_v2z * Tx;
    V = V + PQ * k_Dy;
    W = W + PQ * k_v3y;

    PQ = k_v2y * Tx - k_v2x * Ty;
    V = V + PQ * k_Dz;
    W = W + PQ * k_v3z;

    V = V * DT;
    W = W * DT;

    // The number of hits is very low - atomic functions should not cause many collisions
    if (U >= 0 && V >= 0 && (U + V) <= 1 && W > 0 && W <= 1) // Intersect condition
    {
        unsigned int ind = atomicInc(&no_hit, BLOCK_SIZE_RayMesh); // Local hit counter for current block
        Wshared[ind] = W;                                          // Store all hits of current block in shared memory
        Windex[ind] = i_mesh;                                      // Index of the hit
        ind = atomicInc(&hit_cnt[i_ray], UINT_MAX);                // Global hit counter for all rays
        if (ind < no_hit_W)                                        // Store first hits in global memory
            Wout[i_ray * no_hit_W + ind] = W;
    }

    // Synchronize to make sure all results are ready before writing the output to global memory
    __syncthreads();

    // Thread 0 of each block processes the partial result and writes it to global memory
    float Wtx = 1;        // Minimun
    float Wty = 0;        // Maximum
    unsigned int Wtz = 0; // Index
    if (threadIdx.x == 0 && no_hit > 0)
    {
        for (unsigned int i = 0; i < no_hit; i++)
        {
            Wtx = (Wshared[i] < Wtx) ? Wshared[i] : Wtx;
            Wty = (Wshared[i] > Wty) ? Wshared[i] : Wty;
            Wtz = (Wshared[i] == Wtx) ? Windex[i] : Wtz;
        }
        Wx[gridDim.x * blockIdx.y + i_ray] = Wtx;
        Wy[gridDim.x * blockIdx.y + i_ray] = Wty;
        Wz[gridDim.x * blockIdx.y + i_ray] = Wtz;
    }
}

// KERNEL: Calculate 3D FBS and LBS positions, accumulate per-block-results
__global__ void CalcIntersectPoints(unsigned int no_mesh_blk, float *Ox, float *Oy, float *Oz,
                                    float *Dx, float *Dy, float *Dz, float *Wx, float *Wy, unsigned int *Wz, unsigned int *iFBS)
{
    unsigned int i_ray = blockIdx.x * blockDim.x + threadIdx.x; // Ray index
    unsigned int n_ray = gridDim.x * 32;                        // Number of rays

    float Wtx; // Minimun
    float Wty; // Maximum
    float W_min = 1;
    float W_max = 0;
    unsigned int W_ind = 0;

    // Read first and last interaction point and number of interactions
    for (unsigned int i = 0; i < no_mesh_blk; i++)
    {
        Wty = Wy[i * n_ray + i_ray]; // Global memory read maximum
        if (Wty > 0)
        {
            Wtx = Wx[i * n_ray + i_ray]; // Global memory read minimum
            W_min = (Wtx < W_min) ? Wtx : W_min;
            W_max = (Wty > W_max) ? Wty : W_max;
            W_ind = (Wtx == W_min) ? Wz[i * n_ray + i_ray] + 1 : W_ind; // Global memory read index
        }
        Wy[i * n_ray + i_ray] = 0; // Reset for next chunk
    }

    // Calculate FBS and LBS - overwrite orig and dest in global memory
    float k_Ox = Ox[i_ray];
    float k_Dx = Dx[i_ray];
    Ox[i_ray] = k_Ox + W_min * k_Dx;

    float k_Oy = Oy[i_ray];
    float k_Dy = Dy[i_ray];
    Oy[i_ray] = k_Oy + W_min * k_Dy;

    float k_Oz = Oz[i_ray];
    float k_Dz = Dz[i_ray];
    Oz[i_ray] = k_Oz + W_min * k_Dz;

    Dx[i_ray] = k_Ox + W_max * k_Dx;
    Dy[i_ray] = k_Oy + W_max * k_Dy;
    Dz[i_ray] = k_Oz + W_max * k_Dz;

    iFBS[i_ray] = W_ind;
}


// FUNCTION: Calculate ray-mesh interset points
void ray_mesh_intersect_CUDA(Matrix orig, Matrix dest, Matrix mesh, Matrix fbs, Matrix lbs,
                             unsigned int *hit, unsigned int *iFBS, Matrix Wout, int verbose)
{
    // Input variables:
    //  orig            Ray origin, matrix of size [no_ray x 3]
    //  dest            Ray destinations, matrix of size [no_ray x 3]
    //  mesh            Mesh coordinates (v1 and e12, e13), matrix of size [no_mesh x 9]
    //  verbose         Enables or disbles progress report

    // Output variables:
    //  fbs             First interaction point of the ray with the mesh, matrix of size [no_ray x 3]
    //  lbs             Last interaction point of the ray with the mesh, matrix of size [no_ray x 3]
    //  hit             Number of mesh indersections, vector of length [no_ray]
    //  iFBS            Index of the first mesh element that was hit by the ray, vector of length [no_ray]
    //  Wout            List of normalized distances, values between 0 and 1, matrix of size  [no_hit_W x no_ray]

    // Check inputs
    if (orig.width != 3 || dest.width != 3 || fbs.width != 3 || lbs.width != 3)
    {
        cout << "Error: 'orig', 'dest', 'fbs' and 'lbs' must have 3 columns!" << endl;
        return;
    }
    if (dest.height != orig.height || fbs.height != orig.height || lbs.height != orig.height || Wout.width != orig.height)
    {
        cout << "Error: 'orig', 'dest', 'fbs', 'lbs' and 'Wout' must have the same number of rays!" << endl;
        return;
    }
    if (mesh.width != 9)
    {
        cout << "Error: 'mesh' must have 9 rows!" << endl;
        return;
    }

    // choose which GPU to run on
    gpuErrchk(hipSetDevice(0));

    // Set scheduler to use less CPU  for busy wait loop
    gpuErrchk(hipSetDeviceFlags(hipDeviceScheduleBlockingSync));

    // Determine availabe memory on device
    size_t free;                                                      // Free device memory in byte
    size_t total;                                                     // Total device memory in byte
    gpuErrchk(hipMemGetInfo(&free, &total));                         // Get device memory info
    float freeGB = floor(((float)free) / 1024 / 1024 / 102.4) / 10;   // Free device memory in GB
    float totalGB = floor(((float)total) / 1024 / 1024 / 102.4) / 10; // Total device memory in GB
    float device_mem = freeGB - 0.3;                                  // Used device memory in GB (max. 16 GB)
    device_mem = (device_mem > 15.9) ? 15.9 : device_mem;

    // Progress report
    chrono::steady_clock::time_point begin = chrono::steady_clock::now();
    float vb_dots = 50;
    float m0 = 0;
    if (verbose == 1)
        cout << "LOS det. GPU [" << flush;
    else if (verbose == 2)
        cout << "Avail. memory:  " << freeGB << " GB of " << totalGB << " GB, using max. " << device_mem << " GB" << endl;

    unsigned int no_ray = orig.height;                                                // Number of rays, must be the same in orig and dest
    unsigned int no_ray_d = (no_ray / 32 + 1) * 32;                                   // Number of rays on device, multiple of 32 (warp-size)
    unsigned int no_hit_W = Wout.height;                                              // Number of interactions to be retunrned in the output
    unsigned int no_mesh = mesh.height;                                               // Mesh size
    unsigned int no_mesh_d = (no_mesh / BLOCK_SIZE_RayMesh + 1) * BLOCK_SIZE_RayMesh; // Mesh size on device
    unsigned int no_mesh_blk = no_mesh_d / BLOCK_SIZE_RayMesh;                        // Number of mesh-blocks

    device_mem = (device_mem - 0.1) * 256 * 1024 * 1024;                                    // Number of 32 bit words on device
    unsigned int mem_size = (unsigned int)device_mem;                                       // Number of 32 bit words as unsigned int
    mem_size = mem_size - 9 * no_mesh_d;                                                    // Subtract size of the mesh
    unsigned int no_ray_chnk = (mem_size / (8 + 3 * no_mesh_blk + no_hit_W) / 32 + 1) * 32; // Number of rays to be processed in parallel
    no_ray_chnk = (no_ray_chnk > no_ray_d) ? no_ray_d : no_ray_chnk;                        // Limit to no_ray_d if only one chunk
    unsigned int no_chnk = (unsigned int)ceil((float)no_ray_d / (float)no_ray_chnk);        // Number of chunks

    // Calculate the needed device memory
    mem_size = 9 * no_mesh_d + no_ray_chnk * (8 + 3 * no_mesh_blk + no_hit_W); // 32 Bit words
    device_mem = ceil(4 * (float)mem_size / 1024 / 1024);                      // MB

    if (verbose == 2)
    {
        cout << "Used memory:    " << device_mem << " MB" << endl;
        cout << "Mesh size:      " << no_mesh << " (" << no_mesh_d << ", " << no_mesh_blk << " blocks)" << endl;
        cout << "Number of rays: " << no_ray << " (" << no_ray_d << ")" << endl;
        cout << "Chunk size:     " << no_ray_chnk << " (" << no_chnk << " chunks)" << endl;
    }

    // Allocate device memory
    float *d_v1x, *d_v1y, *d_v1z, *d_v2x, *d_v2y, *d_v2z, *d_v3x, *d_v3y, *d_v3z; // Vertices
    float *d_Ox, *d_Oy, *d_Oz, *d_Dx, *d_Dy, *d_Dz;                               // Rays
    float *d_Wx, *d_Wy;                                                           // Internal storage for partial results
    unsigned int *d_Wz;                                                           // Internal storage for index
    unsigned int *d_hit_cnt;                                                      // Counter for the number of hits
    unsigned int *d_iFBS;                                                         // Index of first hit
    float *d_Wout;                                                                // Normalized intersection coordinates

    size_t size_vert_h = no_mesh * sizeof(float);   // Mesh size on host
    size_t size_vert_d = no_mesh_d * sizeof(float); // Mesh size on device
    size_t size_ray = no_ray_chnk * sizeof(float);  // Ray size
    size_t size_W = no_ray_chnk * no_mesh_blk * sizeof(float);
    size_t size_Wz = no_ray_chnk * no_mesh_blk * sizeof(unsigned int);
    size_t size_cnt = no_ray_chnk * sizeof(unsigned int);
    size_t size_Wout = no_hit_W * no_ray_chnk * sizeof(float);

    gpuErrchk(hipMalloc(&d_v1x, size_vert_d));
    gpuErrchk(hipMalloc(&d_v1y, size_vert_d));
    gpuErrchk(hipMalloc(&d_v1z, size_vert_d));
    gpuErrchk(hipMalloc(&d_v2x, size_vert_d));
    gpuErrchk(hipMalloc(&d_v2y, size_vert_d));
    gpuErrchk(hipMalloc(&d_v2z, size_vert_d));
    gpuErrchk(hipMalloc(&d_v3x, size_vert_d));
    gpuErrchk(hipMalloc(&d_v3y, size_vert_d));
    gpuErrchk(hipMalloc(&d_v3z, size_vert_d));
    gpuErrchk(hipMalloc(&d_Ox, size_ray));
    gpuErrchk(hipMalloc(&d_Oy, size_ray));
    gpuErrchk(hipMalloc(&d_Oz, size_ray));
    gpuErrchk(hipMalloc(&d_Dx, size_ray));
    gpuErrchk(hipMalloc(&d_Dy, size_ray));
    gpuErrchk(hipMalloc(&d_Dz, size_ray));
    gpuErrchk(hipMalloc(&d_Wx, size_W));
    gpuErrchk(hipMalloc(&d_Wy, size_W));
    gpuErrchk(hipMalloc(&d_Wz, size_Wz));
    gpuErrchk(hipMalloc(&d_hit_cnt, size_cnt));
    gpuErrchk(hipMalloc(&d_iFBS, size_cnt));
    gpuErrchk(hipMalloc(&d_Wout, size_Wout));

    // Transfer Mesh to device
    gpuErrchk(hipMemcpy(d_v1x, &mesh.elements[0], size_vert_h, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v1y, &mesh.elements[no_mesh], size_vert_h, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v1z, &mesh.elements[2 * no_mesh], size_vert_h, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v2x, &mesh.elements[3 * no_mesh], size_vert_h, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v2y, &mesh.elements[4 * no_mesh], size_vert_h, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v2z, &mesh.elements[5 * no_mesh], size_vert_h, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v3x, &mesh.elements[6 * no_mesh], size_vert_h, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v3y, &mesh.elements[7 * no_mesh], size_vert_h, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_v3z, &mesh.elements[8 * no_mesh], size_vert_h, hipMemcpyHostToDevice));

    // Calculate the edges V1 to V2 and V1 to V3
    DestMinusOrig<<<no_mesh_d / 32, 32>>>(d_v1x, d_v1y, d_v1z, d_v2x, d_v2y, d_v2z);
    gpuErrchk(hipPeekAtLastError());
    DestMinusOrig<<<no_mesh_d / 32, 32>>>(d_v1x, d_v1y, d_v1z, d_v3x, d_v3y, d_v3z);
    gpuErrchk(hipPeekAtLastError());

    // Process data chunk-wise
    size_t size_current_chunk;
    for (unsigned int chnkIdx = 0; chnkIdx < no_chnk; chnkIdx++)
    {
        // The last chunk is smaller
        size_current_chunk = (chnkIdx == no_chnk - 1) ? (no_ray - chnkIdx * no_ray_chnk) * sizeof(float) : size_ray;

        if (verbose == 2)
            cout << "Processing:     Chunk " << chnkIdx << " with " << size_current_chunk / 4 << " rays." << endl;

        // Transfer ray origins and destinations to device
        gpuErrchk(hipMemcpy(d_Ox, &orig.elements[chnkIdx * no_ray_chnk], size_current_chunk, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_Oy, &orig.elements[chnkIdx * no_ray_chnk + no_ray], size_current_chunk, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_Oz, &orig.elements[chnkIdx * no_ray_chnk + 2 * no_ray], size_current_chunk, hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_Dx, &dest.elements[chnkIdx * no_ray_chnk], size_current_chunk, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_Dy, &dest.elements[chnkIdx * no_ray_chnk + no_ray], size_current_chunk, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(d_Dz, &dest.elements[chnkIdx * no_ray_chnk + 2 * no_ray], size_current_chunk, hipMemcpyHostToDevice));

        // Calculate dest-orig on device
        DestMinusOrig<<<no_ray_chnk / 32, 32>>>(d_Ox, d_Oy, d_Oz, d_Dx, d_Dy, d_Dz);
        gpuErrchk(hipPeekAtLastError());

        // Reset hit-counter on device
        gpuErrchk(hipMemset(d_hit_cnt, 0, size_cnt));
        gpuErrchk(hipMemset(d_Wout, 0, size_Wout));

        // Determine Ray-Mesh intersections
        dim3 dimGrid(no_ray_chnk, no_mesh_blk);
        RayMeshIntersect<<<dimGrid, BLOCK_SIZE_RayMesh>>>(d_v1x, d_v1y, d_v1z, d_v2x, d_v2y, d_v2z, d_v3x, d_v3y, d_v3z,
                                                          d_Ox, d_Oy, d_Oz, d_Dx, d_Dy, d_Dz, d_Wx, d_Wy, d_Wz, d_hit_cnt, d_Wout, no_hit_W);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        // Calculate first and last intersect points
        CalcIntersectPoints<<<no_ray_chnk / 32, 32>>>(no_mesh_blk, d_Ox, d_Oy, d_Oz, d_Dx, d_Dy, d_Dz, d_Wx, d_Wy, d_Wz, d_iFBS);
        gpuErrchk(hipPeekAtLastError());

        // Read FBS and LBS from device
        gpuErrchk(hipMemcpy(&fbs.elements[chnkIdx * no_ray_chnk], d_Ox, size_current_chunk, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(&fbs.elements[chnkIdx * no_ray_chnk + no_ray], d_Oy, size_current_chunk, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(&fbs.elements[chnkIdx * no_ray_chnk + 2 * no_ray], d_Oz, size_current_chunk, hipMemcpyDeviceToHost));

        gpuErrchk(hipMemcpy(&lbs.elements[chnkIdx * no_ray_chnk], d_Dx, size_current_chunk, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(&lbs.elements[chnkIdx * no_ray_chnk + no_ray], d_Dy, size_current_chunk, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(&lbs.elements[chnkIdx * no_ray_chnk + 2 * no_ray], d_Dz, size_current_chunk, hipMemcpyDeviceToHost));

        // Read hit counter and iFBS from device
        size_current_chunk = (chnkIdx == no_chnk - 1) ? (no_ray - chnkIdx * no_ray_chnk) * sizeof(unsigned int) : size_cnt;
        gpuErrchk(hipMemcpy(&hit[chnkIdx * no_ray_chnk], d_hit_cnt, size_current_chunk, hipMemcpyDeviceToHost));
        gpuErrchk(hipMemcpy(&iFBS[chnkIdx * no_ray_chnk], d_iFBS, size_current_chunk, hipMemcpyDeviceToHost));

        // Read normalized intersect distances from device
        size_current_chunk = (chnkIdx == no_chnk - 1) ? no_hit_W * (no_ray - chnkIdx * no_ray_chnk) * sizeof(float) : size_Wout;
        gpuErrchk(hipMemcpy(&Wout.elements[no_hit_W * chnkIdx * no_ray_chnk], d_Wout, size_current_chunk, hipMemcpyDeviceToHost));

        // Update progress bar
        if (verbose == 1)
        {
            float m1 = ceil(vb_dots * ((float)chnkIdx + 1) / ((float)no_chnk));
            if (m1 > m0)
            {
                for (float m2 = 0; m2 < m1 - m0; m2++)
                    cout << "o" << flush;
                m0 = m1;
            }
        }
    }

    // Free device memory
    gpuErrchk(hipFree(d_v1x));
    gpuErrchk(hipFree(d_v1y));
    gpuErrchk(hipFree(d_v1z));
    gpuErrchk(hipFree(d_v2x));
    gpuErrchk(hipFree(d_v2y));
    gpuErrchk(hipFree(d_v2z));
    gpuErrchk(hipFree(d_v3x));
    gpuErrchk(hipFree(d_v3y));
    gpuErrchk(hipFree(d_v3z));
    gpuErrchk(hipFree(d_Ox));
    gpuErrchk(hipFree(d_Oy));
    gpuErrchk(hipFree(d_Oz));
    gpuErrchk(hipFree(d_Dx));
    gpuErrchk(hipFree(d_Dy));
    gpuErrchk(hipFree(d_Dz));
    gpuErrchk(hipFree(d_Wx));
    gpuErrchk(hipFree(d_Wy));
    gpuErrchk(hipFree(d_Wz));
    gpuErrchk(hipFree(d_hit_cnt));
    gpuErrchk(hipFree(d_iFBS));
    gpuErrchk(hipFree(d_Wout));

    gpuErrchk(hipDeviceReset());

    chrono::steady_clock::time_point end = chrono::steady_clock::now();
    if (verbose == 1)
        cout << "] " << chrono::duration_cast<chrono::seconds>(end - begin).count() << " seconds" << endl;
    else if (verbose == 2)
        cout << "Elapsed time:   " << chrono::duration_cast<chrono::seconds>(end - begin).count() << " seconds" << endl;

    return;
}
