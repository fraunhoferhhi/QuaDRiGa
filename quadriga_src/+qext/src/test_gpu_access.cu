#include "hip/hip_runtime.h"
#include "test_gpu_access.h"

// KERNEL: Add two numbers
__global__ void Add_A_and_B(float *d_a, float *d_b, float *d_c)
{
    d_c[0] = d_a[0] + d_b[0];
}

void test_gpu_access_CUDA(double *cc)
{
    cc[0] = 0;         // Initialize output to 0
    hipError_t error; // Initialie CUDA Error

    error = hipSetDevice(0); // choose which GPU to run on
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return;
    }

    float *h_a = new float[0];
    h_a[0] = 3;

    float *h_b = new float[0];
    h_b[0] = 7;

    float *d_a, *d_b, *d_c;
    size_t sz = sizeof(float);

    error = hipMalloc(&d_a, sz);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return;
    }

    error = hipMalloc(&d_b, sz);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return;
    }

    error = hipMalloc(&d_c, sz);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return;
    }

    error = hipMemcpy(d_a, h_a, sz, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return;
    }

    error = hipMemcpy(d_b, h_b, sz, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return;
    }

    Add_A_and_B<<<1, 1>>>(d_a, d_b, d_c);
    error = hipPeekAtLastError();
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return;
    }
    error = hipDeviceSynchronize();
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return;
    }

    float *h_c = new float[0];
    error = hipMemcpy(h_c, d_c, sz, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return;
    }

    if (h_c[0] != 10)
    {
        hipDeviceReset();
        return;
    }

    int *val = new int[0];
    error = hipDeviceGetAttribute(val, hipDeviceAttributeComputeCapabilityMinor, 0);
    if (error != hipSuccess)
    {
        hipDeviceReset();
        return;
    }

    cc[0] = (double)val[0];
    error = hipDeviceGetAttribute(val, hipDeviceAttributeComputeCapabilityMajor, 0);
    cc[0] = (cc[0]) / 10 + (double)val[0];

    hipDeviceReset();
    return;
}
